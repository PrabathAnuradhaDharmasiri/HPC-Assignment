#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    printf("Device Name: %s\n", prop.name);
    printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
    printf("Max thread dimensions (x,y,z): (%d, %d, %d)\n",
           prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("Max grid size (x,y,z): (%d, %d, %d)\n",
           prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);

    int maxThreads = prop.maxThreadsPerBlock * prop.maxGridSize[0];
    printf("👉 Total max threads in one launch (1D grid): %d\n", maxThreads);

    return 0;
}
